#include "hip/hip_runtime.h"
#include <map>
#include <string>

#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/extrema.h>
#include <thrust/remove.h>

#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>
#if CUDA_42
#include <cutil_inline.h>
#endif
#include <cfloat>
#include <time.h>

#include "HSWO.h"
#include "HSWODevice.h"

#include <sstream>
#include "Libraries/Logger.h"

#ifdef LOG_ENABLED
	#define stream_to_log(theStream) {stream_to_log_inner(theStream)}
	#define stream_to_log_sameLine(theStream) {stream_to_log_inner_sameLine(theStream)}
#else
	#define stream_to_log(theStream)
	#define stream_to_log_sameLine(theStream)
#endif

#define stream_to_log_inner(theStream) { using namespace HyperSpectralToolbox; \
					auto sp_ofstream = Logger::openStream(); \
					(*sp_ofstream) << theStream << "\n";	\
					sp_ofstream->close(); }					
			
#define stream_to_log_inner_sameLine(theStream) { using namespace HyperSpectralToolbox; \
					auto sp_ofstream = Logger::openStream(); \
					(*sp_ofstream) << theStream;	\
					sp_ofstream->close(); }	

__global__ void atmoic_exp(int* mutex)
{
	int indexi = blockIdx.x * blockDim.x + threadIdx.x;
	int indexj = blockIdx.y * blockDim.y + threadIdx.y;

	for (int i=0; i < 32; i++) {
		for (int j=0; j < 32; j++) {
			if ((indexi % 32 == i) && (indexj % 32 == j)) {

				while(atomicCAS(mutex,0, 1) == 1)
				{
					printf("Thread %d, %d - cas = %d\n", indexi, indexj, 1);
					//printf("Thread %d - cas = %d\n", indexi, 1);
				}

				printf("Thread %d, %d - inside section \n", indexi, indexj);
				//printf("Thread %d - inside section\n", indexi);

				atomicExch(mutex, 0 );	

			}		
		}
	}
	
}
namespace HyperSpectralToolbox
{
	void run_expr()
	{	
		//mutex initialization (for solution2)
		//int N = 64;
		printf( "expr atomic =============\n" );
		int* dev_mutex2;
		hipMalloc((void**)&dev_mutex2, sizeof(int));		
		hipMemset(dev_mutex2, 0, sizeof(int));

		// Kernel invocation 
		dim3 numBlocks2(2,2); //
		//dim3 numBlocks2(3,1); //

		dim3 threadsPerBlock2(8, 4);  //
		//dim3 threadsPerBlock2(32, 1);  //
		atmoic_exp<<<numBlocks2 , threadsPerBlock2>>>(dev_mutex2);

		hipDeviceSynchronize();
		printf("kernel finished\n");
		hipFree( dev_mutex2 ); //for solution 2
	}
}

//#define MY_FLT_MAX         3.402823466e+38F        /* max value */
//#define MY_FLT_MAX         (3.402823466*1000000.0f)        /* max value */
#define MY_FLT_MAX         (FLT_MAX)        /* max value */

__global__ void kernel_compute_dissim(int* keysFilter ,int* adj, int nRegions, const int nBands
	, float* regionsSums, int* regionsPixelsCount
	, float* regionsAdjMinimums
	, int* regionsAdjMinimumsLabels
	, int* needsAdjRecomputation) 
{ 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	//printf("[Thread %d]: blockIdx.x = %d, blockDim.x = %d, threadIdx.x = %d: Hi ! \n", index, blockIdx.x , blockDim.x , threadIdx.x);

	if(index < nRegions)
	{
		int regionID = keysFilter[index]-1;
		if (regionID == -2) return;

		float SHregionsAdjMinimums;
		SHregionsAdjMinimums = MY_FLT_MAX;
		int SHregionsAdjMinimumsLabels;
		SHregionsAdjMinimumsLabels = -1;

		//printf(" - [Thread %d] region %d\n", index, regionID+1 );

		// ****** Optimization ****************
		if(needsAdjRecomputation[regionID] == 1)
		//if(true)
		{
			///////////////////////////////////////
			int offset_regionID = regionID*MAX_REGION_ADJ;	
			float r1nPixels = regionsPixelsCount[regionID];

			//printf(" - [Thread %d] region %d pixel count = %f \n", index, regionID+1 , r1nPixels);

			for(int a=0; a<MAX_REGION_ADJ; a++)
			{				
				int regionIDadj = adj[offset_regionID+a]-1;				

				if (regionIDadj <= -1) 
				{
					break;
				}
				//if(regionID > regionIDadj) continue;  //optimization			
				float sum_eculidean = 0.0f;
				float r2nPixels = regionsPixelsCount[regionIDadj];
				
				//printf(" - [Thread %d] ADJ region %d pixel count = %f \n", index, regionIDadj+1, r2nPixels);

				float temp;

				int band = 0;

				for (; band < nBands; band++)
				{
					temp = regionsSums[regionID*nBands+band]/(r1nPixels) - regionsSums[regionIDadj*nBands+band]/(r2nPixels);
					//					printf("	--- [Thread %d] region %d sum at band %d = %f \n", index, regionID, band, regionsSums[regionID*nBands+band]);
					//					printf("	--- [Thread %d] region %d sum at band %d = %f \n", index, regionIDadj, band, regionsSums[regionIDadj*nBands+band]);
					sum_eculidean += temp * temp;
				}

				//output[offset_regionID+a] = sqrtf(sum_eculidean) / float(nBands);
				//output[offset_regionID+a] = sqrtf(sum_eculidean*((r1nPixels*r2nPixels)/(r1nPixels+r2nPixels)));
				
				temp = sqrtf(sum_eculidean*((r1nPixels*r2nPixels)/(r1nPixels+r2nPixels)));
				//printf("temp = %f\n", temp);

				if(temp < SHregionsAdjMinimums)  
				{
					SHregionsAdjMinimums = temp;
					SHregionsAdjMinimumsLabels = regionIDadj+1;
					//printf(" - [Thread %d] region %d min distance to neighbor %d = %f \n", index, regionID+1, regionsAdjMinimumsLabels[regionID], regionsAdjMinimums[regionID]);
				}				

				//if(temp < regionsAdjMinimums[regionID])  
				//{
					//regionsAdjMinimums[regionID] = temp;
					//regionsAdjMinimumsLabels[regionID] = regionIDadj+1;
					//printf(" - [Thread %d] region %d min distance to neighbor %d = %f \n", index, regionID+1, regionsAdjMinimumsLabels[regionID], regionsAdjMinimums[regionID]);
				//}				

			}
			//}
			regionsAdjMinimums[regionID] = SHregionsAdjMinimums;
			regionsAdjMinimumsLabels[regionID] = SHregionsAdjMinimumsLabels;
			needsAdjRecomputation[regionID] = 0;
		}
		else
		{
			//no recomputation needed: do nothing
		}
	}
} 

#if ONE_DIMENSIONAL_SPEC_KERNEL
__global__ void kernel_compute_regions_dissim(const int* keysFilter ,const int* adj, int nRegions, const int nBands
	, const float* regionsSums, const int* regionsPixelsCount
	, float* regionsMinimums
	, int* regionsMinimumsLabels
	, int* needsRecomputation) 
{ 
	int index1 = blockIdx.x * blockDim.x + threadIdx.x;	
	int canCompute = 1;

	//if(index1 < nRegions)
	//{
		int regionID1 = keysFilter[index1]-1;
		if (regionID1 == -2) 
		{
			canCompute = 0;
		}

		//__shared__ float r1nPixels;		
		float r1nPixels = regionsPixelsCount[regionID1];
		float SHregionsMinimums = MY_FLT_MAX;
		int SHregionsMinimumsLabels = -1;		

		// ****** Optimization ****************
		//if(true)			
		//if(needsRecomputation[regionID1] == 1)			
		if(canCompute)
		{
			for(int i=0; i < nRegions; ++i)
			{
				int regionID2 = keysFilter[i]-1;		
				int canComputePair = 1;
				if (regionID2 == -2)
				{
					canComputePair = 0;
				}

				if (regionID1 == regionID2)
				{
					canComputePair = 0;
				}

				//__syncthreads();  

				if(canComputePair)
				{
					int offset_regionID = regionID1*MAX_REGION_ADJ;	
					int foundAdjacent = 0;
					//int runFlag = 1;
					//__syncthreads(); 
					for(int a=0; a<MAX_REGION_ADJ; ++a)
					{
						//if(runFlag)
						//{
							int regionIDadj = adj[offset_regionID+a]-1;				

							if (regionIDadj <= -1) 
							{
								break;
								//runFlag = 0;
							}
							if(regionIDadj == regionID2) {
								foundAdjacent = 1;	//after adding this line, the kernel slowed down to 489 seconds!
								// seems pretty nasty thread divergence happned !!
								// Fixed: by adding the __syncthreads(); line after the loop
								break;	
								//runFlag = 0;
							}
						//}
					}

					__syncthreads();  //The fix to thread divergence problem

					if(foundAdjacent == 0)
					{
						float r2nPixels = regionsPixelsCount[regionID2];

						//if(regionID > regionIDadj) continue;  //optimization			
						float sum_dissim = 0.0f;
						float temp;
						for (int band = 0; band < nBands; band++)
						{
							temp = regionsSums[regionID1*nBands+band]/(r1nPixels) - regionsSums[regionID2*nBands+band]/(r2nPixels);
							sum_dissim += temp * temp;
						}

						temp = sqrtf(sum_dissim*((r1nPixels*r2nPixels)/(r1nPixels+r2nPixels)));

						if(temp < SHregionsMinimums) 
						{
							SHregionsMinimums = temp;
							SHregionsMinimumsLabels = regionID2+1;
						}
					}
					
				}				
				//if(temp < regionsMinimums[regionID1]) 
				//{
				//	regionsMinimums[regionID1] = temp;
				//	regionsMinimumsLabels[regionID1] = regionID2+1;
				//}				
			}	
			//__syncthreads();
			regionsMinimums[regionID1] = SHregionsMinimums;
			regionsMinimumsLabels[regionID1] = SHregionsMinimumsLabels;
			needsRecomputation[regionID1] = 0;
		}		
	//}
} 
#endif

__global__ void printTestArr(int* testArr)
{		
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	//for(int t=0; t< 1024;++t)
	printf("[%d] = %d\n", index,testArr[index]);
}

#define BLOCK_BANDS 16
#define BLOCK_WIDTH 16

__global__ void kernel_compute_regions_dissim_2dim(/*int *testArr,*/ const int* keysFilter ,const int* adj, int nRegions, const int nBands
	, const float* regionsSums, const int* regionsPixelsCount
	/*
	, float* regionsMinimums
	, int* regionsMinimumsLabels
	*/
	, int* needsRecomputation	
	, float* _temp_all_dissims) 
{ 
	int region1Index = blockIdx.y * blockDim.y + threadIdx.y;
	int region2Index = blockIdx.x * blockDim.x + threadIdx.x;
	

	//__shared__ int shared_regions1_pixels_count[BLOCK_WIDTH];
	//__shared__ int shared_regions2_pixels_count[BLOCK_WIDTH];

	__shared__ float shared_regions1_sums[BLOCK_WIDTH][BLOCK_BANDS];
	__shared__ float shared_regions2_sums[BLOCK_WIDTH][BLOCK_BANDS];
	if(threadIdx.x == 0) 
	{
		//shared_regions1_pixels_count[threadIdx.y] = regionsPixelsCount[region1Index];
		for(int b=0; b<BLOCK_BANDS; ++b)
		{
			shared_regions1_sums[threadIdx.y][b] = regionsSums[region1Index*nBands+b];
		}
	}

	if(threadIdx.y == 0) 
	{
		//shared_regions2_pixels_count[threadIdx.x] = regionsPixelsCount[region2Index];
		for(int b=0; b<BLOCK_BANDS; ++b)
		{
			shared_regions2_sums[threadIdx.x][b] = regionsSums[region2Index*nBands+b];
		}
	}
	
	__syncthreads();


	float sum_dissim = 0.0f;
	float temp = MY_FLT_MAX;
	int can_compute = 1;

	int region1ID = keysFilter[region1Index]-1;				
	if (region1ID == -2) 
	{
		can_compute = 0;
	}

#if SPECTRAL_DYNAMIC_PROGRAMMING
	if( region1ID != -2)
	{
		if( needsRecomputation[region1ID] == 0 )
		{
			can_compute = 0;
		}
	}
#endif

	int region2ID = keysFilter[region2Index]-1;				
	if (region2ID == -2) 
	{
		can_compute = 0;
	}

	if (region1ID == region2ID) 
	{
		can_compute = 0;
	}	

	//__syncthreads();

	if(can_compute)
	{
		float r1nPixels = regionsPixelsCount[region1ID];
		//float r1nPixels = shared_regions1_pixels_count[threadIdx.y];

		// ****** Optimization ****************			
		//if(needsRecomputation[regionID1] == 1)			
		int offset_regionID = region1ID*MAX_REGION_ADJ;	
		int foundAdjacent = 0;
		for(int a=0; a<MAX_REGION_ADJ; ++a)
		{				
			int regionIDadj = adj[offset_regionID+a]-1;				

			if (regionIDadj <= -1) 
			{
				break;
			}
			if(regionIDadj == region2ID) {						
				foundAdjacent = 1;
				break;
			}
		}

		__syncthreads();

		if(foundAdjacent == 0)
		{
			float r2nPixels = regionsPixelsCount[region2ID];
			//float r2nPixels = shared_regions2_pixels_count[threadIdx.x];

			//if(region1ID > region2ID) return;  //optimization			
			int band = 0;
			for (band = 0; band < BLOCK_BANDS; ++band)
			{
				temp = (shared_regions1_sums[threadIdx.y][band])/(r1nPixels) - (shared_regions2_sums[threadIdx.x][band])/(r2nPixels);
				sum_dissim += temp * temp;
			}

			for (band = band; band < nBands; ++band)
			{
				temp = regionsSums[region1ID*nBands+band]/(r1nPixels) - regionsSums[region2ID*nBands+band]/(r2nPixels);
				sum_dissim += temp * temp;
			}


			temp = sqrtf(sum_dissim*((r1nPixels*r2nPixels)/(r1nPixels+r2nPixels)));

			//if(temp < LocalregionsMinimums) 
			{
				//LocalregionsMinimums = temp;
				//LocalregionsMinimumsLabels = region2ID+1;
				//local_minimums_store[double_idx.local[0]][double_idx.local[1]] = temp;
				//local_minimums_store_labels[double_idx.local[0]][double_idx.local[1]] = region2ID+1;

			}			
		}

		_temp_all_dissims[region1ID*nRegions + region2ID] = temp;	//if region1, region2 pair is to be computed but adjacent to each other, temp will equal MY_FLT_MAX
																	//and the old value in _temp_all_dissims will be rest. (this is a must for correctness)
																	//This solves the changing adjacents problem with _temp_all_dissims array
		//if(temp < regionsMinimums[regionID1]) 
		//{
		//	regionsMinimums[regionID1] = temp;
		//	regionsMinimumsLabels[regionID1] = regionID2+1;
		//}				
		//}			


		//__syncthreads(); //adding this syncthreads increased the time by 4 seconds, so I removed it
		//race (global minimums update)

		////if(region1ID >= 0)
		////{
		//	//mutex		(for CUDA, must serialize the wraps (using these loops and ifs) to avoid deadlock caused by wrap scheduler when he detects path divergence)
		//	//int indexi = blockIdx.x * blockDim.x + threadIdx.x;
		//	//int indexj = blockIdx.y * blockDim.y + threadIdx.y;
		//	
		//	for (int i=0; i < 32; ++i) {
		//		for (int j=0; j < 32; ++j) {
		//			//if ((indexi % 32 == i) && (indexj % 32 == j)) {
		//			if ((region1Index % 32 == i) && (region2Index % 32 == j)) {

		//				//while(atomicCAS(mutex,0, 1) == 1)
		//				//{
		//				//};

		//				///printf("Thread %d, %d - inside section \n", indexi, indexj);
		//				//critical section
		//				//__threadfence();
		//				//testing if wrap threads are serialized or not
		//				//testArr[region1ID] = testArr[region1ID] + 1;

		//				float glbl = regionsMinimums[region1ID] ;

		//				//if(local_minimums_final[double_idx.local[0]] < glbl)
		//				if(temp < glbl)
		//				{
		//					regionsMinimums[region1ID] = temp;
		//					regionsMinimumsLabels[region1ID] = region2ID+1;
		//					//needsRecomputation[regionID1] = 0;
		//				}

		//				//end critical section		
		//				//atomicExch(mutex, 0 );
		//			}
		//		}
		//	}
		////}
	}
	

} 

__global__ void update_all_spectral_minimums(const int maxRegions, const float* _temp_all_dissims, float* regionsMinimums, int* regionsMinimumsLabels)
{
	int regionIndex = blockIdx.x * blockDim.x + threadIdx.x;
	
	for(int i=0; i<maxRegions; ++i)
	{
		float val = _temp_all_dissims[regionIndex*maxRegions+i];
		if(val < regionsMinimums[regionIndex])  
		{
			regionsMinimums[regionIndex] = val;
			regionsMinimumsLabels[regionIndex] = i+1;			
		}
	}

}

__global__ void recompute_others_best_region_dissimilarity_to(int theNewMergedRegionLabel, const int nMaxRegions, int* keysFilter
	, int* needsRecomputation, int* regionsMinimumsLabels, float* regionsMinimums, const int nBands
	, float* regionsSums, int* regionsPixelsCount) 
{ 
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int canCompute = 1;

	if(index < nMaxRegions)
	{
		int regionID = keysFilter[index]-1;
		int theNewMergedRegionID = theNewMergedRegionLabel-1;
		if(regionID == -2) 
		{
			canCompute = 0;
		}
		if(regionID == (theNewMergedRegionID))
		{
			canCompute = 0;
		}

		__syncthreads();
		if(canCompute == 1)
		{
			if(needsRecomputation[regionID] == 0) //compute only for the "no need for computaion" regions
			{
				float temp; float sum_distance=0.0;

				float r_nPixels = regionsPixelsCount[regionID];
				float theNewMergedRegionNPixels = regionsPixelsCount[theNewMergedRegionID];

				for (int band = 0; band < nBands; band++)
				{
					temp = regionsSums[regionID*nBands+band]/(r_nPixels) - regionsSums[theNewMergedRegionID*nBands+band]/(theNewMergedRegionNPixels);
					sum_distance += temp * temp;
				}

				temp = sqrtf(sum_distance*((r_nPixels*theNewMergedRegionNPixels)/(r_nPixels+theNewMergedRegionNPixels)));

				if(temp < regionsMinimums[regionID])  
				{
					regionsMinimums[regionID] = temp;
					regionsMinimumsLabels[regionID] = theNewMergedRegionLabel;			
				}
			}
		}
	}
}


//__global__ void kernel_compute_regions_dissim(int* keysFilter ,int* adj, int nRegions, const int nBands
//	, float* regionsSums, float* output, int* regionsPixelsCount
//	, float* regionsMinimums
//	, int* regionsMinimumsLabels
//	, int* needsRecomputation
//	, float lastMinDissim
//	, float spclustWeight) 
//{ 
//	int index1 = blockIdx.x * blockDim.x + threadIdx.x;
//	int index2 = blockIdx.y * blockDim.y + threadIdx.y;
//	
//	if(index1 < nRegions && index2 < nRegions)
//	{
//		int regionID1 = keysFilter[index1]-1;
//		int regionID2 = keysFilter[index2]-1;
//		if (regionID1 == -2) return;
//		if (regionID2 == -2) return;
//		if (regionID1 == regionID2) return;
//
//		int offset_regionID = regionID1*MAX_REGION_ADJ;	
//		for(int a=0; a<MAX_REGION_ADJ; a++)
//		{				
//				int regionIDadj = adj[offset_regionID+a]-1;				
//
//				if (regionIDadj <= -1) 
//				{
//					break;
//				}
//				if(regionIDadj == regionID2) return;
//		}
//		//printf(" - [Thread %d] region %d\n", index, regionID+1 );
//
//		// ****** Optimization ****************
//		//if(needsAdjRecomputation[regionID] == 1)
//		if(true)
//		{
//			///////////////////////////////////////
//			float r1nPixels = regionsPixelsCount[regionID1];
//
//			//printf(" - [Thread %d] region %d pixel count = %f \n", index, regionID+1 , r1nPixels);
//
//			//if(regionID > regionIDadj) continue;  //optimization			
//			float sum_dissim = 0.0f;
//			float r2nPixels = regionsPixelsCount[regionID2];
//
//			//printf(" - [Thread %d] ADJ region %d pixel count = %f \n", index, regionIDadj+1, r2nPixels);
//
//			float temp; int band = 0;
//
//			for (; band < nBands; band++)
//			{
//				temp = regionsSums[regionID1*nBands+band]/(r1nPixels) - regionsSums[regionID2*nBands+band]/(r2nPixels);
//				sum_dissim += temp * temp;
//			}
//
//			temp = sqrtf(sum_dissim*((r1nPixels*r2nPixels)/(r1nPixels+r2nPixels)));
//			//printf("temp = %f\n", temp);
//
//			if(temp < regionsMinimums[regionID1] && temp < (lastMinDissim*spclustWeight) ) 
//			{
//				regionsMinimums[regionID1] = temp;
//				regionsMinimumsLabels[regionID1] = regionID2+1;
//				//printf(" - [Thread %d] region %d min distance to neighbor %d = %f \n", index, regionID+1, regionsAdjMinimumsLabels[regionID], regionsAdjMinimums[regionID]);
//			}				
//
//
//			needsRecomputation[regionID1] = 0;
//		}
//		else
//		{
//			//no recomputation needed: do nothing
//		}
//	}
//} 

__global__ void reset_others_best_region_computation_flags_from(int region1Label, int region2Label, const int nMaxRegions, int* keysFilter
	, int* needsRecomputation, int* regionsMinimumsLabels, float* regionsMinimums)  
{ 
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if(index < nMaxRegions)
	{
		int regionID = keysFilter[index]-1;
		if(regionID == -2) return;
		if(regionsMinimumsLabels[regionID] == region1Label || regionsMinimumsLabels[regionID] == region2Label) 
		{	
			needsRecomputation[regionID] = 1;
			regionsMinimumsLabels[regionID] = -1;
			regionsMinimums[regionID] = MY_FLT_MAX;			
		}
	}
}

__global__ void kernel_fill_empty_dissims(const int nMaxRegions, float* output) 
{ 
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if(index < nMaxRegions)
	{
		int regionID = index;
		int offset_regionID = regionID*MAX_REGION_ADJ;	

		for(int a=0; a<MAX_REGION_ADJ; a++)
		{
			output[offset_regionID+a] = MY_FLT_MAX;
		}		
	}
}

template <typename T>
__global__ void kernelInitializeArray(T* __restrict__ a, const T value, const size_t n) {
      int tid = threadIdx.x + blockDim.x * blockIdx.x;
      if (tid < n) {
           a[tid] = value;
       }
}

namespace HyperSpectralToolbox
{
	void DeviceInitializeData(HSWO* object)
	{	
		//if (hipDeviceSetCacheConfig(hipFuncCachePreferEqual) != hipSuccess)
		{
			//printf("failed config\n");
		}

		// Allocate whole data cube in device memory
		int N = object->m_image_width*object->m_image_height*object->m_nBands;
		object->m_deviceData.m_dev_nRegions = object->m_image_width*object->m_image_height;
		
		//size_t size = N*sizeof(float);
		//CUDA_SAFE_CALL( hipMalloc((void**)&object->m_deviceData.m_dev_pDataCube, size) );
		
		// Copy data cube from host memory to device memory
		//CUDA_SAFE_CALL( hipMemcpy(object->m_deviceData.m_dev_pDataCube, object->m_pDataCube.get(), size, hipMemcpyHostToDevice) );
		
		//regions keys
		size_t regionKeysSize = object->m_image_height*object->m_image_width*sizeof(int);
		hipMalloc((void**)&(object->m_deviceData.m_dev_RegionKeys), regionKeysSize);		
		
		
		//adjacents
		size_t regionAdjSize = object->m_image_height*object->m_image_width*MAX_REGION_ADJ*sizeof(int);
		hipMalloc((void**)&(object->m_deviceData.m_dev_pRegionAdjancencies), regionAdjSize);		
		
		
		//means
		int nMeans = object->m_image_height*object->m_image_width *object->m_nBands;
		
		size_t regionMeans = nMeans * sizeof(float);
		hipMalloc((void**)&(object->m_deviceData.m_dev_pRegionMeans), regionMeans);
		
		
		//TODO can be removed, I now don't use the dissims ....
		//allocate device memory for regions dissimilarities 
		//size_t regionDissimsAdjSize = object->m_image_height*object->m_image_width*MAX_REGION_ADJ*sizeof(float);
		//CUDA_SAFE_CALL( hipMalloc((void**)&(object->m_deviceData.m_dev_pDissims), regionDissimsAdjSize) );

		//regions pixel count
		{
			size_t regionPixelCountsSize = object->m_image_height*object->m_image_width*sizeof(int);
			hipMalloc((void**)&(object->m_deviceData.m_dev_RegionPixelsCount), regionPixelCountsSize);
		}

		//m_dev_regionsAdjMinimums
		size_t regionsAdjMinimums = object->m_image_height*object->m_image_width;
		hipMalloc((void**)&(object->m_deviceData.m_dev_regionsAdjMinimums), regionsAdjMinimums*sizeof(float));
		//m_dev_regionsAdjMinimumsLabels
		hipMalloc((void**)&(object->m_deviceData.m_dev_regionsAdjMinimumsLabels), regionsAdjMinimums*sizeof(int)) ;
		//m_dev_needsAdjRecomputation
		hipMalloc((void**)&(object->m_deviceData.m_dev_needsAdjRecomputation), regionsAdjMinimums*sizeof(int)) ;

		//m_dev_regionsMinimums		
		hipMalloc((void**)&(object->m_deviceData.m_dev_regionsMinimums), regionsAdjMinimums*sizeof(float)) ;
		//m_dev_regionsMinimumsLabels
		hipMalloc((void**)&(object->m_deviceData.m_dev_regionsMinimumsLabels), regionsAdjMinimums*sizeof(int)) ;
		//m_dev_needsRecomputation
		hipMalloc((void**)&(object->m_deviceData.m_dev_needsRecomputation), regionsAdjMinimums*sizeof(int)) ;

		//new, for the 2dim spectral kernel
		size_t temp_all_dissims_size = object->m_deviceData._temp_all_dissims_size * sizeof(float);
		CUDA_SAFE_CALL (hipMalloc((void**)&(object->m_deviceData._temp_all_dissims), temp_all_dissims_size); );
		
#if SPECTRAL_DYNAMIC_PROGRAMMING
		//int	threadsPerBlock = object->m_deviceData.m_threadsPerBlock;
		//int number_of_blocks =  object->m_deviceData._temp_all_dissims_size / (threadsPerBlock*threadsPerBlock);
		//dim3 gridDim(number_of_blocks, 1);
		//dim3 blockDim(threadsPerBlock*threadsPerBlock, 1);
		//kernelInitializeArray<float> <<<gridDim, blockDim>>>(object->m_deviceData._temp_all_dissims, MY_FLT_MAX, object->m_deviceData._temp_all_dissims_size);
		//hipDeviceSynchronize();
#endif
		printf("temp_all_dissims_size = %d\n",temp_all_dissims_size);


	}
	
	void DeviceExit(HSWO* object)
	{
		// Free used device memory
		//if(object->m_deviceData.m_dev_pDataCube != NULL) 
		//{
			//CUDA_SAFE_CALL( hipFree(object->m_deviceData.m_dev_pDataCube) );;			
		//}
		//object->m_deviceData.m_dev_pDataCube = NULL;
		
		//========
		if(object->m_deviceData.m_dev_pRegionMeans != NULL) 
		{
			CUDA_SAFE_CALL( hipFree(object->m_deviceData.m_dev_pRegionMeans) );;			
		}
		object->m_deviceData.m_dev_pRegionMeans = NULL;
		
		//========
		//if(object->m_deviceData.m_dev_pDissims != NULL) 
		//{
		//		CUDA_SAFE_CALL( hipFree(object->m_deviceData.m_dev_pDissims) );;			
		//}
		//object->m_deviceData.m_dev_pDissims = NULL;
		
		//========
		if(object->m_deviceData.m_dev_pRegionAdjancencies != NULL) 
		{
			CUDA_SAFE_CALL( hipFree(object->m_deviceData.m_dev_pRegionAdjancencies) );;			
		}
		object->m_deviceData.m_dev_pRegionAdjancencies = NULL;
		
		//========
		if(object->m_deviceData.m_dev_RegionKeys != NULL) 
		{
			CUDA_SAFE_CALL( hipFree(object->m_deviceData.m_dev_RegionKeys) );;			
		}
		object->m_deviceData.m_dev_RegionKeys = NULL;

		//========
		if(object->m_deviceData.m_dev_RegionPixelsCount != NULL) 
		{
			CUDA_SAFE_CALL( hipFree(object->m_deviceData.m_dev_RegionPixelsCount) );;			
		}
		object->m_deviceData.m_dev_RegionPixelsCount = NULL;

		//========		
		{
			if(object->m_deviceData.m_dev_regionsAdjMinimums != NULL) 
			{
				CUDA_SAFE_CALL( hipFree(object->m_deviceData.m_dev_regionsAdjMinimums) );;			
			}
			object->m_deviceData.m_dev_regionsAdjMinimums = NULL;

			if(object->m_deviceData.m_dev_regionsAdjMinimumsLabels != NULL) 
			{
				CUDA_SAFE_CALL( hipFree(object->m_deviceData.m_dev_regionsAdjMinimumsLabels) );;			
			}
			object->m_deviceData.m_dev_regionsAdjMinimumsLabels = NULL;

			if(object->m_deviceData.m_dev_needsAdjRecomputation != NULL) 
			{
				CUDA_SAFE_CALL( hipFree(object->m_deviceData.m_dev_needsAdjRecomputation) );;			
			}
			object->m_deviceData.m_dev_needsAdjRecomputation = NULL;
		}

		//========		
		{
			if(object->m_deviceData.m_dev_regionsMinimums != NULL) 
			{
				CUDA_SAFE_CALL( hipFree(object->m_deviceData.m_dev_regionsMinimums) );;			
			}
			object->m_deviceData.m_dev_regionsMinimums = NULL;

			if(object->m_deviceData.m_dev_regionsMinimumsLabels != NULL) 
			{
				CUDA_SAFE_CALL( hipFree(object->m_deviceData.m_dev_regionsMinimumsLabels) );;			
			}
			object->m_deviceData.m_dev_regionsMinimumsLabels = NULL;

			if(object->m_deviceData.m_dev_needsRecomputation != NULL) 
			{
				CUDA_SAFE_CALL( hipFree(object->m_deviceData.m_dev_needsRecomputation) );;			
			}
			object->m_deviceData.m_dev_needsRecomputation = NULL;
		}

		//=====
		if(object->m_deviceData._temp_all_dissims != NULL) 
		{
			CUDA_SAFE_CALL( hipFree(object->m_deviceData._temp_all_dissims) );;			
		}
		object->m_deviceData._temp_all_dissims = NULL;
	}
	
	void DeviceInitStep(HSWO* object)
	{
		//copy the following data to device
		//1- region adjacencies
		//2- region means
		//3- keys (regionID filter)
		
		//size_t regionKeysSize = object->m_image_height*object->m_image_width*sizeof(int);
		//hipMemcpy(object->m_deviceData.m_dev_RegionKeys, object->m_deviceData.m_pRegionKeys, regionKeysSize , hipMemcpyHostToDevice) ;
		
		//hipError_t err = hipGetLastError(); 
		//std::string serr = std::string(hipGetErrorString(err)); 		
	}

	void DeviceUploadInitialMeans(HSWO* object)
	{
		//========================================================================
		//upload the means data				
		/*
		//int nMeans = object->m_image_height*object->m_image_width *object->m_nBands;
		//size_t regionMeans = nMeans * sizeof(float);
		hipMemcpy(object->m_deviceData.m_dev_pRegionMeans,
			object->m_deviceData.m_pRegionsMeans,
			regionMeans, hipMemcpyHostToDevice) ;
			*/
	
		std::hash_map<int, HSWO::Region*>::iterator it = object->m_regions.begin();
		for (;it != object->m_regions.end(); it++)
		{			
			int currentRegionLabel = it->first;
 			int device_pointer_offset = (currentRegionLabel-1)*object->m_nBands; 			
			hipMemcpy(object->m_deviceData.m_dev_pRegionMeans+device_pointer_offset,
				it->second->sumOfPixels,
				object->m_nBands*sizeof(float), hipMemcpyHostToDevice) ;


			hipError_t err = hipGetLastError(); 
			if(err != hipError_t::hipSuccess) {std::cout << std::string(hipGetErrorString(err)) << std::endl;}
		}
	}
	
	void DeviceUploadInitialAdjacents(HSWO* object)
	{
		//upload the initial adjaccents data		
		size_t regionAdjSize = object->m_image_height*object->m_image_width*MAX_REGION_ADJ*sizeof(int);
		hipMemcpy(object->m_deviceData.m_dev_pRegionAdjancencies, object->m_deviceData.m_pRegionAdjancencies
			, regionAdjSize, hipMemcpyHostToDevice) ;
		
		hipError_t err = hipGetLastError(); 
		if(err != hipError_t::hipSuccess) {std::cout << std::string(hipGetErrorString(err)) << std::endl;}
	}	

	void DeviceUpdateRegionMean(HSWO* object, int lastChangedRegionLabel, HSWO::Region* ptrLastChangedRegion)
	{
		int nMeans = object->m_nBands;
		size_t regionMeans = nMeans * sizeof(float);
		int device_pointer_offset = (lastChangedRegionLabel-1)*object->m_nBands; 
		
		/*
		hipMemcpy(object->m_deviceData.m_dev_pRegionMeans+device_pointer_offset,
			&(object->m_deviceData.m_pRegionsMeans[(lastChangedRegionLabel-1)*object->m_nBands]),
			regionMeans, hipMemcpyHostToDevice) ;
			*/

		hipMemcpyAsync(object->m_deviceData.m_dev_pRegionMeans+device_pointer_offset,
			ptrLastChangedRegion->sumOfPixels,
			regionMeans, hipMemcpyHostToDevice) ;

		hipError_t err = hipGetLastError(); 
		if(err != hipError_t::hipSuccess) {std::cout << std::string(hipGetErrorString(err)) << std::endl;}
	}

	void DeviceExitStep(HSWO* object)
	{
	}
	
	typedef thrust::tuple<float,int> Tuple; 
	struct min_index 
	{ 
		__host__ __device__ 
			Tuple operator()(Tuple a, Tuple b) 
		{ 
			if (thrust::get<0>(a) < thrust::get<0>(b)) 
				return a; 
			else 
				return b; 
		} 
	}; 

	void DeviceCalcAllDissims(HSWO* object, unsigned int threadsPerBlock, int& label1, int& label2, float& minDissim)
	{
		int N = object->_max_nRegions;
		//int N = object->m_deviceData.m_regions.size();
		int maxRegions = object->_max_nRegions;
		
		//emptying all dissims values from previous computation
		//for
		//thrust::device_ptr<float> pdev1(object->m_deviceData.m_dev_pDissims);
		//thrust::fill_n(pdev1, maxRegions*MAX_REGION_ADJ, MY_FLT_MAX);
		
		/*dim3 numBlocks1(256,1);
		int threadsForEveryBlock1 = 256;
		dim3 threadsPerBlock1(threadsForEveryBlock1, 1); 
		kernel_fill_empty_dissims<<<numBlocks1, threadsPerBlock1>>>(maxRegions,object->m_deviceData.m_dev_pDissims);
		*/

		// Kernel invocation 
		//int threadsForEveryBlock2 = threadsPerBlock;
		int threadsForEveryBlock2 = HARD_CODED_1_DIM_CUDA_KERNELS_BLOCK_WIDTH;	
		//printf("threads per block = %d \n", threadsForEveryBlock2);

		int nBlocks = N/threadsForEveryBlock2;
		dim3 numBlocks2(nBlocks,1);
		//printf("n blocks = %d \n", numBlocks2.x);
		
		dim3 threadsPerBlock2(threadsForEveryBlock2, 1); 
		//kernel_compute_dissim<<<numBlocks2, threadsPerBlock2, object->m_nBands*threadsForEveryBlock2* sizeof(float) >>>(object->m_deviceData.m_dev_RegionKeys
		kernel_compute_dissim<<<numBlocks2, threadsPerBlock2>>>(object->m_deviceData.m_dev_RegionKeys
															  ,object->m_deviceData.m_dev_pRegionAdjancencies
															  ,N		
															  ,object->m_nBands
															  ,object->m_deviceData.m_dev_pRegionMeans															  
															  ,object->m_deviceData.m_dev_RegionPixelsCount
															  ,object->m_deviceData.m_dev_regionsAdjMinimums
															  ,object->m_deviceData.m_dev_regionsAdjMinimumsLabels
															  ,object->m_deviceData.m_dev_needsAdjRecomputation); 
		
		
#if CUDA_42
		hipDeviceSynchronize();
#else
		hipDeviceSynchronize();
#endif

		//printf("done kernel\n");
		
		// reduction step: get the minimum dissim measure and its region pair IDs
		// so that we can merge them on the host		
		//thrust::device_ptr<float> pdev(object->m_deviceData.m_dev_pDissims);
		//int maxDissims = maxRegions*MAX_REGION_ADJ;
		//thrust::device_ptr<float> result = thrust::min_element(pdev,pdev+(maxDissims) );
		//int index = (result - pdev);	

		thrust::device_ptr<float> pdev(object->m_deviceData.m_dev_regionsAdjMinimums);
		int maxDissims = maxRegions;
		
		thrust::device_ptr<float> result = thrust::min_element(pdev,pdev+(maxDissims) );
		minDissim = *result;		
		
		label1 = (result - pdev) + 1;		
		hipMemcpy(&label2, object->m_deviceData.m_dev_regionsAdjMinimumsLabels+(label1-1), sizeof(int), hipMemcpyDeviceToHost);
		

		//thrust::counting_iterator<int> Y(0);  
		//Tuple init(pdev[0],Y[0]); 
		//Tuple result = thrust::reduce 
		//	(thrust::make_zip_iterator(thrust::make_tuple(pdev, Y)), 
		//	thrust::make_zip_iterator(thrust::make_tuple(pdev+(maxDissims),   Y +(maxDissims))), 
		//	init, 
		//	min_index());  
		//float value; int index;  thrust::tie(value,index) = result; 
 
		////int index = result - object->m_deviceData.m_dev_pDissims;
				
		object->_last_merge_dissim_value = minDissim;
		object->_last_block_count = nBlocks;
		
		//return index;
		
	}

	
	void DeviceCalcRegionDissims(HSWO* object, unsigned int threadsPerBlock, int& label1, int& label2, float& minDissim)
	{
		int N = object->_max_nRegions;
		int maxRegions = object->_max_nRegions;

		
		// Kernel invocation parameters
		int threadsForEveryBlock2 = threadsPerBlock;	
		int nBlocks = N/threadsForEveryBlock2;
#if ONE_DIMENSIONAL_SPEC_KERNEL
		dim3 numBlocks2(nBlocks,1);
		dim3 threadsPerBlock2(threadsForEveryBlock2, 1); 
#else		
		//int *testArr = NULL;

		//hipMalloc((void**)&testArr, 1024*sizeof(int));		
		//hipMemset(testArr, 0, 1024*sizeof(int));
		
		
		//mutex initialization (for solution2)
		//int* dev_mutex2;
		//hipMalloc((void**)&dev_mutex2, sizeof(int));		
		//hipMemset(dev_mutex2, 0, sizeof(int));

		dim3 numBlocks2(nBlocks,nBlocks); //for solution 2
		dim3 threadsPerBlock2(threadsForEveryBlock2, threadsForEveryBlock2);  //for solution 2
#endif		
		
		//clock_t startClock2 = clock();

#if ONE_DIMENSIONAL_SPEC_KERNEL
#else
		//emptying all dissims values from previous computation		
		int number_of_blocks =  object->m_deviceData._temp_all_dissims_size / (threadsPerBlock*threadsPerBlock);
		dim3 gridDim(number_of_blocks, 1);
		dim3 blockDim(threadsPerBlock*threadsPerBlock, 1);
		kernelInitializeArray<float> <<<gridDim, blockDim>>>(object->m_deviceData._temp_all_dissims, MY_FLT_MAX, object->m_deviceData._temp_all_dissims_size);
		hipDeviceSynchronize();

#endif
		//clock_t endClock2 = clock();
		//HSWO::time_count += (endClock2 - startClock2);
		
		//clock_t startClock2 = clock();

#if ONE_DIMENSIONAL_SPEC_KERNEL
		kernel_compute_regions_dissim<<<numBlocks2, threadsPerBlock2>>>(object->m_deviceData.m_dev_RegionKeys
																,object->m_deviceData.m_dev_pRegionAdjancencies
																,N		
																,object->m_nBands
																,object->m_deviceData.m_dev_pRegionMeans
																,object->m_deviceData.m_dev_RegionPixelsCount														  
																,object->m_deviceData.m_dev_regionsMinimums
																,object->m_deviceData.m_dev_regionsMinimumsLabels
																,object->m_deviceData.m_dev_needsRecomputation															  
																);
#else
		kernel_compute_regions_dissim_2dim<<<numBlocks2, threadsPerBlock2>>>(/*testArr, */object->m_deviceData.m_dev_RegionKeys //for solution 2		
															  ,object->m_deviceData.m_dev_pRegionAdjancencies
															  ,N		
															  ,object->m_nBands
															  ,object->m_deviceData.m_dev_pRegionMeans
															  ,object->m_deviceData.m_dev_RegionPixelsCount
															  /*
															  ,object->m_deviceData.m_dev_regionsMinimums
															  ,object->m_deviceData.m_dev_regionsMinimumsLabels															  
															  */
															  ,object->m_deviceData.m_dev_needsRecomputation															  
															  ,object->m_deviceData._temp_all_dissims); 
#endif
		
#if CUDA_42
		hipDeviceSynchronize();
#else
		hipDeviceSynchronize();
#endif
		//printf("\nreached here 2\n");

		//clock_t endClock2 = clock();
		//HSWO::time_count += (endClock2 - startClock2);

		//printf("Test=======================\n");
		//printTestArr<<< 1024/128, 128>>>(testArr);

		//// First reduction
		//int number_of_blocks_reduction_1 =  object->_max_nRegions / (HARD_CODED_1_DIM_CUDA_KERNELS_BLOCK_WIDTH);
		//dim3 gridDim_reduction_1(number_of_blocks_reduction_1, 1);
		//dim3 blockDim_reduction_1(HARD_CODED_1_DIM_CUDA_KERNELS_BLOCK_WIDTH, 1);
		//update_all_spectral_minimums<<< gridDim_reduction_1, blockDim_reduction_1>>>(object->_max_nRegions
		//													,object->m_deviceData._temp_all_dissims
		//													,object->m_deviceData.m_dev_regionsMinimums
		//													,object->m_deviceData.m_dev_regionsMinimumsLabels);
		//hipDeviceSynchronize();

		////Final reduction
		//thrust::device_ptr<float> pdev(object->m_deviceData.m_dev_regionsMinimums);
		//int maxDissims = maxRegions;

		//thrust::device_ptr<float> result = thrust::min_element(pdev,pdev+(maxDissims) );
		//label1 = (result - pdev) + 1;		
		//hipMemcpy(&label2, object->m_deviceData.m_dev_regionsMinimumsLabels+(label1-1), sizeof(int), hipMemcpyDeviceToHost);
		//minDissim = *result;

#if ONE_DIMENSIONAL_SPEC_KERNEL
		//Final reduction
		thrust::device_ptr<float> pdev(object->m_deviceData.m_dev_regionsMinimums);
		int maxDissims = maxRegions;

		thrust::device_ptr<float> result = thrust::min_element(pdev,pdev+(maxDissims) );
		label1 = (result - pdev) + 1;		
		hipMemcpy(&label2, object->m_deviceData.m_dev_regionsMinimumsLabels+(label1-1), sizeof(int), hipMemcpyDeviceToHost);
		minDissim = *result;

#else

		thrust::device_ptr<float> pdev(object->m_deviceData._temp_all_dissims);
		int maxDissims = object->m_deviceData._temp_all_dissims_size;

		thrust::device_ptr<float> result = thrust::min_element(pdev,pdev+(maxDissims) );
		int offset = (result - pdev);		
		label1 = offset / maxRegions +1;
		label2 = offset % maxRegions +1 ;

		minDissim = *result;

#endif		
		//printf("last mindissim = %f, new dissim = %f, %d, %d\n", object->_last_merge_dissim_value, minDissim , label1, label2);
		//printf("\nreached here 2.2\n");

		if(minDissim < (object->_last_merge_dissim_value*object->_spclustWeight) )
		{
			//printf("found spclust merge\n");			
		}
		else
		{
			minDissim = FLT_MAX;
			label1 = -1;
			label2 = -1;
		}
		object->_last_block_count = nBlocks;

#if ONE_DIMENSIONAL_SPEC_KERNEL
#else
		//printf("\nreached here 2.3\n");
		//hipFree( dev_mutex2 ); //for solution 2
		//hipFree( testArr);
#endif
	}

	void DeviceResetOthersBestRegionComputationFlagsFrom(HSWO* object, int region1Label, int region2Label ) 
	{		
		int maxRegions = object->_max_nRegions;
		
		// Kernel invocation 
		//int threadsForEveryBlock2 = object->m_deviceData.m_threadsPerBlock;	
		int threadsForEveryBlock2 = HARD_CODED_1_DIM_CUDA_KERNELS_BLOCK_WIDTH;	
		int nBlocks = maxRegions/threadsForEveryBlock2;
		dim3 numBlocks2(nBlocks,1);
		
		dim3 threadsPerBlock2(threadsForEveryBlock2, 1); 
		reset_others_best_region_computation_flags_from<<<numBlocks2, threadsPerBlock2>>>(region1Label, region2Label
																, maxRegions
																, object->m_deviceData.m_dev_RegionKeys
															  ,object->m_deviceData.m_dev_needsRecomputation													
															  ,object->m_deviceData.m_dev_regionsMinimumsLabels
															  ,object->m_deviceData.m_dev_regionsMinimums); 
		
		hipDeviceSynchronize();
	}

	void DeviceRecomputeOthersBestRegionsDissimilarityTo(HSWO* object, int regionLabel)
	{		
		int maxRegions = object->_max_nRegions;
		
		// Kernel invocation 
		//int threadsForEveryBlock2 = object->m_deviceData.m_threadsPerBlock;	
		int threadsForEveryBlock2 = HARD_CODED_1_DIM_CUDA_KERNELS_BLOCK_WIDTH;	
		int nBlocks = maxRegions/threadsForEveryBlock2;
		dim3 numBlocks2(nBlocks,1);
		
		dim3 threadsPerBlock2(threadsForEveryBlock2, 1); 
		recompute_others_best_region_dissimilarity_to<<<numBlocks2, threadsPerBlock2>>>(regionLabel
																, maxRegions
																, object->m_deviceData.m_dev_RegionKeys
															  ,object->m_deviceData.m_dev_needsRecomputation													
															  ,object->m_deviceData.m_dev_regionsMinimumsLabels
															  ,object->m_deviceData.m_dev_regionsMinimums
															  ,object->m_nBands
															  ,object->m_deviceData.m_dev_pRegionMeans
															  ,object->m_deviceData.m_dev_RegionPixelsCount);  
		
		hipDeviceSynchronize();
	}

	void DeviceCalcAllRegionsMeanVectors(HSWO* object)
	{
		/*
		//allocate region means matrix of nRegions rows x nBands columns
		// rows : regions
		// columns : nBands columns with each columns value 
		// represent the mean value of the region at this 
		// band
		 
		int N = object->m_dev_nRegions * object->m_nBands;
		size_t size = N*sizeof(float);
		CUDA_SAFE_CALL( hipMalloc((void**)&object->m_dev_pRegionMeans, size) );
		
		//calculate
		
		// Copy data cube from host memory to device memory
		CUDA_SAFE_CALL( hipMemcpy(object->m_dev_pDataCube, object->m_pDataCube, size, hipMemcpyHostToDevice) );
*/		
	}
	
	void DeviceThrustRemove(int* pArray, int size, int valueToRemove)
	{
		thrust::device_ptr<int> pdevArray(pArray);
		thrust::remove(pdevArray, pdevArray+size, valueToRemove);
	}

	//template <typename T> 
	void DeviceThrustFill(int* pdeviceArray, int size, int newValue)
	{ 
		thrust::device_ptr<int> pdevArray(pdeviceArray);
		thrust::fill_n(pdevArray, size, newValue);
	}
	void DeviceThrustFillFloat(float* pdeviceArray, int size, float newValue)
	{ 
		thrust::device_ptr<float> pdevArray(pdeviceArray);
		thrust::fill_n(pdevArray, size, newValue);
	}

	void DeviceThrustResetAdjacencyOfMergedRegion(int* m_deviceDatam_dev_pRegionAdjancencies, int lastMergedRegion_OffsetInMatrix)
	{
		//emptying all adjacencies on the device of the merged region
		thrust::device_ptr<int> pdevArray(m_deviceDatam_dev_pRegionAdjancencies+lastMergedRegion_OffsetInMatrix);
		thrust::fill_n(pdevArray, MAX_REGION_ADJ, -1);
	}

	void DeviceThrustReplace(int* m_deviceData_DOT_m_dev_pRegionAdjancencies_PLUS_currentAdjacentRegion_OffsetInMatrix
		, int nAdjacentsOfAdjacent, int adjRegionLabel, int regionLabel )
	{		
		thrust::device_ptr<int> pdevArray(m_deviceData_DOT_m_dev_pRegionAdjancencies_PLUS_currentAdjacentRegion_OffsetInMatrix);
		thrust::replace(pdevArray, pdevArray+nAdjacentsOfAdjacent, adjRegionLabel, regionLabel);		
	}

	float computeMax(HSWO* object)
	{
		if(object->m_deviceData.m_dev_pDataCube == NULL) exit(0);
		
		int N = object->m_image_width*object->m_image_height*object->m_nBands;
		
		thrust::device_ptr<float> dev_ptr(object->m_deviceData.m_dev_pDataCube);	
		
		thrust::device_ptr<float> result = thrust::max_element(dev_ptr,dev_ptr+N);
		
		return *result;
	}
	
	float computeMin(HSWO* object)
	{
		if(object->m_deviceData.m_dev_pDataCube == NULL) exit(0);
		
		int N = object->m_image_width*object->m_image_height*object->m_nBands;
		
		// Use already copied data to device
		thrust::device_ptr<float> dev_ptr(object->m_deviceData.m_dev_pDataCube);	
		
		thrust::device_ptr<float> result = thrust::min_element(dev_ptr,dev_ptr+N);
		
		return *result;
	}
	
	void runThrustExpr(thrust::host_vector<int>& h_vec, int N)
	{
		// transfer data to the device
		thrust::device_vector<int> d_vec = h_vec;
		
		// sort data on the device
		
		thrust::sort(d_vec.begin(), d_vec.end());
		
		// transfer data back to host
		thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
	}
}